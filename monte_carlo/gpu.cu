#include "hip/hip_runtime.h"
#include <hiprand.h>

#include "gpu.h"

#define SEED 1234567ull

#define simpleRand(seed) \
    { seed = (1103515245ull * seed + 12345ull) % (1ull << 31); }

__global__ void computePi(size_t t, size_t resolution, num radius, size_t *results) {
    num threadid = blockIdx.x * blockDim.x + threadIdx.x;
    num seed = SEED + threadid + t * (10'000'000);

    for (size_t i = 0; i < resolution; ++i) {
        // Draw random x and y
        simpleRand(seed);
        num x = seed % radius;
        simpleRand(seed);
        num y = seed % radius;

        // If the point is within the circle (top right quarter)
        if (x * x + y * y <= RADIUS * RADIUS) ++results[threadid];
    }
}

size_t gpuPi(size_t times, size_t resolution, num radius, size_t blocks,
             size_t threads) {
    size_t cpupi = 0;
    size_t nparallel = blocks * threads;

    size_t *results;
    hipMallocManaged(&results, nparallel * sizeof(size_t));

    // Multiply pi by 4
    times *= 4;

    for (size_t t = 0; t < times; ++t) {
        for (size_t i = 0; i < nparallel; ++i) results[i] = 0;

        computePi<<<blocks, threads>>>(t, resolution, radius, results);
        hipDeviceSynchronize();

        for (size_t i = 0; i < nparallel; ++i) cpupi += results[i];
    }

    hipFree(results);

    return cpupi;
}
